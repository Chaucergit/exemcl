#include "hip/hip_runtime.h"
#include <Eigen/Eigen>
#include <gtest/gtest.h>
#include <src/function/SubmodularFunction.h>
#include <src/function/cpu/ExemplarClusteringSubmodularFunction.h>
#include <src/function/gpu/ExemplarClusteringSubmodularFunction.cuh>
#include <tests/CSVFile.h>

#ifndef EXEMCL_TESTFILES_DIR
#error No testfile directory supplied. Compilation aborted.
#else
std::string TESTFILES_ROOT(EXEMCL_TESTFILES_DIR);
#endif

template<typename HostDataType>
struct SubmodularTestData {
    exemcl::MatrixX<HostDataType> groundSet;
    std::vector<exemcl::MatrixX<HostDataType>> subsets;

    exemcl::VectorX<HostDataType> fValuesExpected;
    exemcl::VectorX<HostDataType> marginalsExpected;

    exemcl::VectorX<HostDataType> marginal;
};

template<typename HostDataType>
SubmodularTestData<HostDataType> loadSubmodularTestData(const std::string& subdir) {
    SubmodularTestData<HostDataType> testData;

    // Load test data.
    testData.groundSet = CSVFile::readCSVFile(TESTFILES_ROOT + subdir + "/ground_set.csv", ',')->asMatrix().cast<HostDataType>();
    exemcl::MatrixX<HostDataType> subsetsMatrix = CSVFile::readCSVFile(TESTFILES_ROOT + subdir + "/subsets.csv", ',')->asMatrix().cast<HostDataType>();
    std::vector<std::vector<exemcl::VectorX<HostDataType>>> subsetVectors;
    for (unsigned int i = 0; i < subsetsMatrix.rows(); i++) {
        exemcl::VectorX<HostDataType> matrixRow = subsetsMatrix.row(i);
        unsigned int subsetIdx = matrixRow[subsetsMatrix.cols() - 1];
        if (subsetIdx >= subsetVectors.size())
            subsetVectors.emplace_back();
        subsetVectors[subsetIdx].push_back(matrixRow.head(matrixRow.rows() - 1));
    }

    // Create subsets.
    for (auto& subset : subsetVectors) {
        exemcl::MatrixX<HostDataType> subsetMatrix(subset.size(), testData.groundSet.cols());
        for (unsigned int i = 0; i < subset.size(); i++)
            subsetMatrix.row(i) = subset[i];
        testData.subsets.push_back(subsetMatrix);
    }

    testData.fValuesExpected = CSVFile::readCSVFile(TESTFILES_ROOT + subdir + "/f_values.csv", ',')->asMatrix().col(0).cast<HostDataType>();
    testData.marginalsExpected = CSVFile::readCSVFile(TESTFILES_ROOT + subdir + "/marginal_values.csv", ',')->asMatrix().col(0).cast<HostDataType>();
    testData.marginal = testData.groundSet.row(testData.groundSet.rows() - 1);

    return testData;
}

template<typename HostDataType>
void testSubmodularFunction(exemcl::SubmodularFunction<HostDataType>& submodularFunction, SubmodularTestData<HostDataType>& testData, HostDataType tolerancy) {
    // Test for correct individual evaluation.
    for (unsigned long i = 0; i < testData.subsets.size(); i++)
        EXPECT_NEAR(testData.fValuesExpected(i), submodularFunction(testData.subsets[i]), tolerancy);

    // Test for correct joint evaluation.
    auto fValuesComputedJoint = submodularFunction(testData.subsets);
    for (unsigned long i = 0; i < testData.subsets.size(); i++)
        EXPECT_NEAR(testData.fValuesExpected(i), fValuesComputedJoint[i], tolerancy);

    // Test for correct individual gains.
    for (unsigned long i = 0; i < testData.subsets.size(); i++)
        EXPECT_NEAR(testData.marginalsExpected(i), submodularFunction(testData.subsets[i], testData.marginal), tolerancy);

    // Test for correct joint gains.
    auto marginalsComputedJoint = submodularFunction(testData.subsets, testData.marginal);
    for (unsigned long i = 0; i < testData.subsets.size(); i++)
        EXPECT_NEAR(testData.marginalsExpected(i), marginalsComputedJoint[i], tolerancy);

    // Test for correct multiple marginals.
    exemcl::MatrixX<HostDataType> emptySet(0, testData.groundSet.cols());
    for (auto& S : testData.subsets) {
        std::vector<HostDataType> individualGains;
        std::vector<exemcl::VectorXRef<HostDataType>> marginalsTested;
        for (unsigned int i = 0; i < S.rows(); i++) {
            exemcl::VectorXRef<HostDataType> marginal = S.row(i);
            individualGains.push_back(submodularFunction(emptySet, marginal));
            marginalsTested.push_back(marginal);
        }

        std::vector<HostDataType> totalGains = submodularFunction(emptySet, marginalsTested);

        EXPECT_EQ(individualGains.size(), totalGains.size());
        for (unsigned int i = 0; i < individualGains.size(); i++)
            EXPECT_NEAR(individualGains[i], totalGains[i], tolerancy);
    }
}

#define FP16_ERROR_TOLERANCY 0.01f
#define FP32_ERROR_TOLERANCY 0.001f
#define FP64_ERROR_TOLERANCY 0.000000000001

using DeviceDataTypes = ::testing::Types<__half, float, double>;
template<typename T>
class GPUTests : public ::testing::Test { };
TYPED_TEST_SUITE(GPUTests, DeviceDataTypes);

TYPED_TEST(GPUTests, ExemplarClusteringST) {
    if constexpr (std::is_same<TypeParam, float>::value || std::is_same<TypeParam, double>::value) {
        // Load test data.
        SubmodularTestData<TypeParam> testData = loadSubmodularTestData<TypeParam>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, TypeParam> submodularFunction(testData.groundSet, 1);

        // Run the test function.
        if constexpr (std::is_same<TypeParam, float>::value)
            testSubmodularFunction(submodularFunction, testData, FP32_ERROR_TOLERANCY);
        else
            testSubmodularFunction(submodularFunction, testData, FP64_ERROR_TOLERANCY);
    } else if constexpr (std::is_same<TypeParam, __half>::value) {
        // Load test data.
        SubmodularTestData<float> testData = loadSubmodularTestData<float>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, float> submodularFunction(testData.groundSet, 1);

        // Run the test function.
        testSubmodularFunction(submodularFunction, testData, FP16_ERROR_TOLERANCY);
    }
}

TYPED_TEST(GPUTests, ExemplarClusteringMT) {
    if constexpr (std::is_same<TypeParam, float>::value || std::is_same<TypeParam, double>::value) {
        // Load test data.
        SubmodularTestData<TypeParam> testData = loadSubmodularTestData<TypeParam>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, TypeParam> submodularFunction(testData.groundSet, -1);

        // Run the test function.
        if constexpr (std::is_same<TypeParam, float>::value)
            testSubmodularFunction(submodularFunction, testData, FP32_ERROR_TOLERANCY);
        else
            testSubmodularFunction(submodularFunction, testData, FP64_ERROR_TOLERANCY);
    } else if constexpr (std::is_same<TypeParam, __half>::value) {
        // Load test data.
        SubmodularTestData<float> testData = loadSubmodularTestData<float>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, float> submodularFunction(testData.groundSet, -1);

        // Run the test function.
        testSubmodularFunction(submodularFunction, testData, FP16_ERROR_TOLERANCY);
    }
}

TYPED_TEST(GPUTests, ExemplarClusteringChunked) {
    if constexpr (std::is_same<TypeParam, float>::value || std::is_same<TypeParam, double>::value) {
        // Load test data.
        SubmodularTestData<TypeParam> testData = loadSubmodularTestData<TypeParam>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, TypeParam> submodularFunction(testData.groundSet, 1);
        submodularFunction.setGPUMemoryLimit(100 * 1024); // 100 KiB

        // Run the test function.
        if constexpr (std::is_same<TypeParam, float>::value)
            testSubmodularFunction(submodularFunction, testData, FP32_ERROR_TOLERANCY);
        else
            testSubmodularFunction(submodularFunction, testData, FP64_ERROR_TOLERANCY);
    } else if constexpr (std::is_same<TypeParam, __half>::value) {
        // Load test data.
        SubmodularTestData<float> testData = loadSubmodularTestData<float>("exem");

        // Create submodular function.
        exemcl::gpu::ExemplarClusteringSubmodularFunction<TypeParam, float> submodularFunction(testData.groundSet, 1);
        submodularFunction.setGPUMemoryLimit(100 * 1024); // 100 KiB

        // Run the test function.
        testSubmodularFunction(submodularFunction, testData, FP16_ERROR_TOLERANCY);
    }
}

using HostDataTypes = ::testing::Types<float, double>;
template<typename T>
class CPUTests : public ::testing::Test { };
TYPED_TEST_SUITE(CPUTests, HostDataTypes);

TYPED_TEST(CPUTests, ExemplarClusteringST) {
    // Load test data.
    SubmodularTestData<TypeParam> testData = loadSubmodularTestData<TypeParam>("exem");

    // Create submodular function.
    exemcl::cpu::ExemplarClusteringSubmodularFunction<TypeParam> submodularFunction(testData.groundSet, 1);

    // Run the test function.
    if constexpr (std::is_same<TypeParam, float>::value)
        testSubmodularFunction(submodularFunction, testData, FP32_ERROR_TOLERANCY);
    else
        testSubmodularFunction(submodularFunction, testData, FP64_ERROR_TOLERANCY);
}

TYPED_TEST(CPUTests, ExemplarClusteringMT) {
    // Load test data.
    SubmodularTestData<TypeParam> testData = loadSubmodularTestData<TypeParam>("exem");

    // Create submodular function.
    exemcl::cpu::ExemplarClusteringSubmodularFunction<TypeParam> submodularFunction(testData.groundSet, -1);

    // Run the test function.
    if constexpr (std::is_same<TypeParam, float>::value)
        testSubmodularFunction(submodularFunction, testData, FP32_ERROR_TOLERANCY);
    else
        testSubmodularFunction(submodularFunction, testData, FP64_ERROR_TOLERANCY);
}

int main(int argc, char** argv) {
    std::cout << "Reading testfiles from: " << TESTFILES_ROOT << std::endl;
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}