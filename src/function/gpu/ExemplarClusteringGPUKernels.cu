#include "hip/hip_runtime.h"
#ifndef EXEMCL_EXEMPLARCLUSTERINGGPUKERNELS_CU
#define EXEMCL_EXEMPLARCLUSTERINGGPUKERNELS_CU

template<typename DeviceDataType>
__global__ void exemplarClusteringKernel(const DeviceDataType* vMatrix, const int nV, const DeviceDataType* summaryMatrix, const int maxS, const int* summarySizes,
                                         const int nS_multi, const int dim, DeviceDataType* resultMatrix) {
    // Create a variable, which represents the current v and S to work on.
    int vJob = blockDim.x * blockIdx.x + threadIdx.x;
    int sJob = blockDim.y * blockIdx.y + threadIdx.y;

    // Check, whether we have a valid V job.
    if (vJob < nV) {
        // Load the current v into shared memory.
        extern __shared__ unsigned char _vShared[];
        auto* vShared = reinterpret_cast<DeviceDataType*>(_vShared);

        if (threadIdx.y == 0) {
            for (int d = 0; d < dim; d++) {
                vShared[threadIdx.x * dim + d] = vMatrix[d * nV + vJob];
            }
        }

        // Ensure, that every block has loaded "their" v vectors into shared memory.
        __syncthreads();

        // Check, whether we have a valid S job.
        if (sJob < nS_multi) {
            // Compute the minimum distance for v and all vectors in S.
            DeviceDataType minDistance = std::numeric_limits<DeviceDataType>::max();

            // Iterate over all vectors in S.
            for (int i = 0; i < summarySizes[sJob]; i++) {
                DeviceDataType distance = 0.0;
                for (int d = 0; d < dim; d++) {
                    if constexpr (std::is_same<DeviceDataType, float>::value)
                        distance += powf(vShared[threadIdx.x * dim + d] - summaryMatrix[i * nS_multi + d * maxS * nS_multi + sJob], 2.0f);
                    else
                        distance += pow(vShared[threadIdx.x * dim + d] - summaryMatrix[i * nS_multi + d * maxS * nS_multi + sJob], 2.0);
                }
                minDistance = minDistance > distance ? distance : minDistance;
            }

            // Write the min distance for this summary into memory.
            resultMatrix[(long) vJob * (long) nS_multi + (long) sJob] = minDistance / (DeviceDataType) nV;
        }
    }
}

__global__ void exemplarClusteringKernel(const __half* vMatrix, const int nV, const __half* summaryMatrix, const int maxS, const int* summarySizes, const int nS_multi,
                                         const int dim, float* resultMatrix) {
#define V_ACCESS(dim_idx) vSharedHalf[threadIdx.x * dim + (dim_idx)]
#define SMAT_ACCESS(dim_idx) summaryMatrix[i * nS_multi + (dim_idx) *maxS * nS_multi + sJob]
    // Create a variable, which represents the current v and S to work on.
    int vJob = blockDim.x * blockIdx.x + threadIdx.x;
    int sJob = blockDim.y * blockIdx.y + threadIdx.y;

    // Check, whether we have a valid V job.
    if (vJob < nV) {
        // Load the current v into shared memory.
        extern __shared__ __half vSharedHalf[];
        if (threadIdx.y == 0) {
            for (int d = 0; d < dim; d++) {
                vSharedHalf[threadIdx.x * dim + d] = vMatrix[d * nV + vJob];
            }
        }

        // Ensure, that every block has loaded "their" v vectors into shared memory.
        __syncthreads();

        // Check, whether we have a valid S job.
        if (sJob < nS_multi) {
            // Compute the minimum distance for v and all vectors in S.
            __half minDistance = 0x7BFF; // fp16 max

            // Iterate over all vectors in S.
            for (int i = 0; i < summarySizes[sJob]; i++) {
                __half2 distance2(0.0, 0.0);
                int d = 0;
                while (d < dim) {
                    __half2 vData(V_ACCESS(d), V_ACCESS(d + 1));
                    __half2 sData(SMAT_ACCESS(d), SMAT_ACCESS(d + 1));
                    __half2 diffData = __hsub2(vData, sData);
                    __half2 powData = __hmul2(diffData, diffData);
                    distance2 += powData;

                    // Increase `d`.
                    d += 2;
                }
                __half distance = distance2.x + distance2.y;
                if (dim % 2 == 1) {
                    __half diff = V_ACCESS(dim - 1) - SMAT_ACCESS(dim - 1);
                    distance += __hmul(diff, diff);
                }

                minDistance = minDistance > distance ? distance : minDistance;

                // Write the min distance for this summary into memory.
                resultMatrix[(long) vJob * (long) nS_multi + (long) sJob] = minDistance / (__half) nV;
            }
        }
    }
}

#endif // EXEMCL_EXEMPLARCLUSTERINGGPUKERNELS_CU
